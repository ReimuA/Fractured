#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>

#include "json.hpp"

#define PI (3.14159)

using json = nlohmann::json;

typedef struct IFSTransform {
    float a;
    float b;
    float c;
    float d;
    float e;
    float f;
}   IFSTransform;

typedef struct WeightedVariation {
    char name[42];
    float weight;
} WeightedVariation;

typedef struct FlamesComponent {
    float weight;
    IFSTransform transform;
    WeightedVariation weightedVariations[5];
} FlamesComponent;

typedef struct Flames {
    int ResolutionX;
    int ResolutionY;
    FlamesComponent finalComponent;
    FlamesComponent components[4];
}   Flames;

__device__
float modn(float a, float b)
{
  int tmp = (int)(a / b);
  return a - ((float) tmp * b);
}				

__device__
void processFlamesComponent(float* x, float* y, FlamesComponent* component) {
    float newX = 0;
    float newY = 0;
    auto t = component->transform;
    float tpx = t.a * (*x) + t.b * (*y) + t.c;
    float tpy = t.d * (*x) + t.e * (*y) + t.f;

    for (auto i = 0; i < 5; i++) {
        auto variation = component->weightedVariations[i];
        auto name = variation.name;
        float vpx = 0;
        float vpy = 0;

        float length = sqrt(tpx * tpx + tpy * tpy);
        float l2 = length * length;


        if (name[0] == 'S') {
            float cosine = cos(l2);
            float sine = sin(l2);
            vpx = tpx * sine - tpy * cosine;
            vpy = tpx * cosine + tpy * sine;
        }

        if (name[0] == 'F') {
            float theta = atan2(py, px);
            float tc2 = PI * t.c * t.c;
            float d = modn(theta + t.f, tc2);

            if (d > 0.5) {
				vpx = length * cos(theta - tc2 / 2);
				vpy = length * sin(theta - tc2 / 2);
			} else {
                vpx = length * cos(theta + tc2 / 2);
                vpy = length * sin(theta + tc2 / 2);
            }
        }

        if (name[0] == 'L') {
            vpx = tpx;
            vpy = tpy;
        }

        newX += vpx * variation.weight;
        newY += vpy * variation.weight;
    }

    *x = newX;
    *y = newY;
}

__global__
void processFlames(int* heatmap, Flames* flames)
{
    hiprandState state;

    hiprand_init(clock() + threadIdx.x, 0, 0, &state);
    float y = hiprand_uniform(&state) * 2 - 1;
    float x = hiprand_uniform(&state) * 2 - 1;

    for (int i = 0; i < 1e7; i++) {
        float r = hiprand_uniform(&state);
        float accumulator = 0;

        FlamesComponent component = flames->components[0];
        for (auto j = 0; j < 4; j++) {
            accumulator += flames->components[j].weight;
            if (accumulator > r) {
                component = flames->components[j];
                break;
            }
        }

        processFlamesComponent(&x, &y, &component);

        if (i > 20) {
            auto px = round((x + 2 * (flames->ResolutionX / flames->ResolutionY)) * (flames->ResolutionY / 4));
            auto py = round((y + 2) * (flames->ResolutionY / 4));

            if (px >= 0 && px < flames->ResolutionX && py >= 0 && py < flames->ResolutionY) {
                int idx = py * flames->ResolutionX + px;
                atomicAdd(&heatmap[idx], 1);
            }
        }
    }
}

void writeResult(int* heatmap, int length) {
    std::ofstream f("res.json");
    if (f.is_open())
    {
        f << "[";
        for (int i = 0; i < length; i++) {
            f << (heatmap[i]);
            if (i != length - 1)
                f << ",";
        }

        f << "]";

        f.close();
    }
    else
        std::cout << "Unable to open file";
}

void variationFromJObject(WeightedVariation* variation, json obj) {
    memset(variation->name, '\0', 42);
    variation->weight = obj["weight"];
    std::string name(obj["variation"]["name"]);
    strcpy(variation->name, name.c_str());
}

void flamesComponentFromJObject(FlamesComponent* component, json obj) {
    component->transform.a = obj["transform"]["a"];
    component->transform.b = obj["transform"]["b"];
    component->transform.c = obj["transform"]["c"];
    component->transform.d = obj["transform"]["d"];
    component->transform.e = obj["transform"]["e"];
    component->transform.f = obj["transform"]["f"];


    std::cout << component->transform.a << std::endl;
    std::cout << component->transform.b << std::endl;
    std::cout << component->transform.c << std::endl;
    std::cout << component->transform.d << std::endl;
    std::cout << component->transform.e << std::endl;
    std::cout << component->transform.f << std::endl;

    component->weight = obj["weight"];

    for (int i = 0; i < 5; i++) {
        variationFromJObject(&component->weightedVariations[i], obj["weightedVariations"].at(i));

        std::cout << component->weightedVariations[i].weight << std::endl;
        std::cout << component->weightedVariations[i].name << std::endl;
    }
}

void readFlames(Flames* flames, char* s) {
    std::string filename(s);
    std::ifstream input(s);
    json data = json::parse(input);

    flames->ResolutionX = data["resolution"]["x"] * 2;
    flames->ResolutionY = data["resolution"]["y"] * 2;

    for (int i = 0; i < 4; i++) {
        flamesComponentFromJObject(&(flames->components[i]), data["components"].at(i));
    }

    flamesComponentFromJObject(&flames->finalComponent, data["final"]);

    std::cout << data["resolution"]["x"] << std::endl;
    std::cout << data["resolution"]["y"] << std::endl;
}


int main(int argc, char** argv)
{
    Flames flames;
    
    if (argc != 2) {
        std::cout << "Usage : ./a.exe flamePath" << std::endl;
        return 1;
    }
    else {
        readFlames(&flames, argv[1]);
    }


    int* heatmap;
    int heatmapLength = flames.ResolutionX * flames.ResolutionY;
    Flames* flamesPtr;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&flamesPtr, sizeof(Flames));

    printf("\n Error msg: %s", hipGetErrorString(hipGetLastError()));
    hipMallocManaged(&heatmap, sizeof(int) * heatmapLength);
    printf("\n Error msg: %s", hipGetErrorString(hipGetLastError()));
    hipMemset(heatmap, 0, heatmapLength * sizeof(int));
    printf("\n Error msg: %s", hipGetErrorString(hipGetLastError()));
    hipMemcpy(flamesPtr, &flames, sizeof(Flames), hipMemcpyHostToDevice);
    printf("\n Error msg: %s", hipGetErrorString(hipGetLastError()));

    processFlames << <1024, 1 >> > (heatmap, flamesPtr);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    printf("\n Error msg: %s", hipGetErrorString(hipGetLastError()));
    writeResult(heatmap, heatmapLength);

    hipFree(heatmap);
    hipFree(flamesPtr);
    return 0;
}