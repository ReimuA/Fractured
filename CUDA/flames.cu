#include "hip/hip_runtime.h"
#include <iostream>
#include <fstream>
#include <math.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <stdlib.h>

#include "json.hpp"

#define PI (3.14159)

using json = nlohmann::json;

typedef struct IFSTransform {
    double a;
    double b;
    double c;
    double d;
    double e;
    double f;
}   IFSTransform;

typedef struct WeightedVariation {
    char name[42];
    double weight;
} WeightedVariation;

typedef struct FlamesComponent {
    double weight;
    IFSTransform transform;
    WeightedVariation weightedVariations[5];
} FlamesComponent;

typedef struct Flames {
    double ResolutionX;
    double ResolutionY;
    FlamesComponent finalComponent;
    FlamesComponent components[4];
}   Flames;

__device__
double modn(double a, double b)
{
  int tmp = (int)(a / b);
  return a - ((double) tmp * b);
}				

__device__
void processFlamesComponent(double* x, double* y, FlamesComponent* component) {
    double newX = 0;
    double newY = 0;
    auto t = component->transform;
    double tpx = t.a * (*x) + t.b * (*y) + t.c;
    double tpy = t.d * (*x) + t.e * (*y) + t.f;

    for (auto i = 0; i < 5; i++) {
        auto variation = component->weightedVariations[i];
        auto name = variation.name;
        double vpx = 0;
        double vpy = 0;

        double length = sqrt(tpx * tpx + tpy * tpy);
        double l2 = length * length;


        if (name[0] == 'S') {
            double cosine = cos(l2);
            double sine = sin(l2);
            vpx = tpx * sine - tpy * cosine;
            vpy = tpx * cosine + tpy * sine;
        }

        if (name[0] == 'F') {
            double theta = atan2(tpy, tpx);
            double tc2 = PI * t.c * t.c;
            double d = modn(theta + t.f, tc2);

            if (d > 0.5) {
				vpx = length * cos(theta - tc2 / 2);
				vpy = length * sin(theta - tc2 / 2);
			} else {
                vpx = length * cos(theta + tc2 / 2);
                vpy = length * sin(theta + tc2 / 2);
            }
        }

        if (name[0] == 'L') {
            vpx = tpx;
            vpy = tpy;
        }

        newX += vpx * variation.weight;
        newY += vpy * variation.weight;
    }

    *x = newX;
    *y = newY;
}

__global__
void processFlames(int* heatmap, Flames* flames)
{
    hiprandState state;

    hiprand_init(clock64() + threadIdx.x, 0, 0, &state);
    double y = hiprand_uniform(&state) * 2 - 1;
    double x = hiprand_uniform(&state) * 2 - 1;

    for (int i = 0; i < 1e6 / 32; i++) {
        double r = hiprand_uniform(&state);
        double accumulator = 0;

        FlamesComponent component = flames->components[0];
        for (auto j = 0; j < 4; j++) {
            accumulator += flames->components[j].weight;
            if (accumulator > r) {
                component = flames->components[j];
                break;
            }
        }

        processFlamesComponent(&x, &y, &component);

        if (i > 20) {
            
				int px = round((x + 2 * (flames->ResolutionX / flames->ResolutionY)) * (flames->ResolutionY / 4.0));
				int py = round((y + 2) * (flames->ResolutionY / 4.0));

			if (px > 0 && px < flames->ResolutionX && py > 0 && py < flames->ResolutionY) {
				int idx = py * (flames->ResolutionX) + px;
				atomicAdd(heatmap + idx, 1);
            }
        }
    }
}

void writeResult(int* heatmap, int length, char *outPath) {
    std::string path(outPath);
    std::ofstream f(path);

    if (f.is_open())
    {
        f << "[";
        for (int i = 0; i < length; i++) {
            f << (heatmap[i]);
            if (i != length - 1)
                f << ",";
        }

        f << "]";

        f.close();
    }
    else
        std::cout << "Unable to open file";
}

void variationFromJObject(WeightedVariation* variation, json obj) {
    memset(variation->name, '\0', 42);
    variation->weight = obj["weight"];
    std::string name(obj["variation"]["name"]);
    strcpy(variation->name, name.c_str());
}

void flamesComponentFromJObject(FlamesComponent* component, json obj) {
    component->transform.a = obj["transform"]["a"];
    component->transform.b = obj["transform"]["b"];
    component->transform.c = obj["transform"]["c"];
    component->transform.d = obj["transform"]["d"];
    component->transform.e = obj["transform"]["e"];
    component->transform.f = obj["transform"]["f"];


    std::cout << component->transform.a << std::endl;
    std::cout << component->transform.b << std::endl;
    std::cout << component->transform.c << std::endl;
    std::cout << component->transform.d << std::endl;
    std::cout << component->transform.e << std::endl;
    std::cout << component->transform.f << std::endl;

    component->weight = obj["weight"];

    for (int i = 0; i < 5; i++) {
        variationFromJObject(&component->weightedVariations[i], obj["weightedVariations"].at(i));

        std::cout << component->weightedVariations[i].weight << std::endl;
        std::cout << component->weightedVariations[i].name << std::endl;
    }
}

void readFlames(Flames* flames, std::string filename) {
    std::ifstream input(filename);
    json data = json::parse(input);

    flames->ResolutionX = data["resolution"]["x"] * 2;
    flames->ResolutionY = data["resolution"]["y"] * 2;

    for (int i = 0; i < 4; i++) {
        flamesComponentFromJObject(&(flames->components[i]), data["components"].at(i));
    }

    flamesComponentFromJObject(&flames->finalComponent, data["final"]);

    std::cout << flames->ResolutionX << std::endl;
    std::cout << flames->ResolutionY << std::endl;
}


int main(int argc, char** argv)
{
    Flames flames;
    
    if (argc != 3) {
        std::cout << "Usage : ./a.exe flamePath outPath" << std::endl;
        return 1;
    } 
    
    std::string flamesMetadata(argv[1]);

    readFlames(&flames, argv[1]);


    int* heatmap;
    int heatmapLength = flames.ResolutionX * flames.ResolutionY;
    Flames* flamesPtr;

    // Allocate Unified Memory – accessible from CPU or GPU
    hipMallocManaged(&flamesPtr, sizeof(Flames));

    printf("\n Error msg: %s", hipGetErrorString(hipGetLastError()));
    hipMallocManaged(&heatmap, sizeof(int) * heatmapLength);
    printf("\n Error msg: %s", hipGetErrorString(hipGetLastError()));
    hipMemset(heatmap, 0, heatmapLength * sizeof(int));
    printf("\n Error msg: %s", hipGetErrorString(hipGetLastError()));
    hipMemcpy(flamesPtr, &flames, sizeof(Flames), hipMemcpyHostToDevice);
    printf("\n Error msg: %s", hipGetErrorString(hipGetLastError()));

    processFlames << <1024, 128 >> > (heatmap, flamesPtr);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    printf("\n Error msg: %s", hipGetErrorString(hipGetLastError()));
    writeResult(heatmap, heatmapLength, argv[2]);

    hipFree(heatmap);
    hipFree(flamesPtr);
    return 0;
}